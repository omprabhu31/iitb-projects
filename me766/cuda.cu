#include <stdlib.h>
#include <stdio.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <time.h>

using namespace std;
#define TILE 16
  
void initialize_matrices(double** a, double** l, double** u, int size)
{
  for (int i = 0; i < size; ++i)
  {
    a[i] = new double[size];
    l[i] = new double[size];
    u[i] = new double[size];
  }
}

__global__ void scaleIndex(double *matrix, int n, int index){
  int start=(index*n+index);
	int end=(index*n+n);
	
	for(int i= start+1 ; i<end; ++i){
		matrix[i]=(matrix[i]/matrix[start]);
	}

}


__global__ void elim(double *A, int n, int index, int bsize){
	extern __shared__ double pivot[];

	int idThread=threadIdx.x;
	int idBlock=blockIdx.x;
	int blockSize=bsize;


	if(idThread==0){
	     for(int i=index;i<n;i++) pivot[i]=A[(index*n)+i];
	}

	__syncthreads();
	int pivotRow=(index*n);
	int currentRow=(((blockSize*idBlock) + idThread)*n);
	int start=currentRow+index;
	int end=currentRow+n;
	if(currentRow >pivotRow){
    for(int i= start+1; i<end; ++i){

        A[i]=A[i]-(A[start]*pivot[i-currentRow]);

             }
      }
}

void fillMatrix(double* a, int n){

   for (int i = 0; i <= (n*n); ++i) {
    a[i] =((rand()%10)+1);
  }


  int diagCount = 0;
  double sum = 0;
  for(int i = 0; i < n; ++i){

    for(int j = i*n; j < i*n + n; ++j){
      sum += abs(a[j]);

    }

    sum -= abs(a[i*n + diagCount]);
    a[i*n + diagCount] = sum + ((rand()%5)+1);
    ++diagCount;
    sum = 0;
  }

}

void printMatrix(double* a, int n){
    for(int i=0; i<(n*n); ++i){
           if(i%n==0)
       		   cout << endl << left << setw(9) << setprecision(3) << a[i] << left <<  setw(9);
           else cout << left << setw(9) << setprecision(3) << a[i] << left <<  setw(9);
         }

    cout << endl;
}

void print2D(double** matrix, int size)
{
  
  for (int i = 0; i < size; i++)
  {
   
    for (int j = 0; j < size; j++)
    {
      
      cout << left << setw(9) << setprecision(3) << matrix[i][j] << left <<  setw(9);
    }
   
    cout << endl;
  }
}

int main(int argc, char** argv){
  
  int n = atoi(argv[1]);

  srand(1);

  
  double *a = new double[n*n];
  double *ret=new double[n*n];

  
  fillMatrix(a, n);
 

  
  double *da;
  int numblock = n/TILE + ((n%TILE)?1:0);

  double runtime;
  runtime = clock()/(double)CLOCKS_PER_SEC;
  hipMalloc(&da, n*n* sizeof (double));
  hipMemcpy(da, a, n*n*sizeof(double), hipMemcpyHostToDevice);
  for(int i=0; i<n; ++i){
      scaleIndex<<<1,1>>>(da,n,i);
	    elim<<<numblock,TILE,n*sizeof(double)>>>(da,n,i,TILE);
   }
  hipMemcpy(ret, da, n*n*sizeof(double),hipMemcpyDeviceToHost );
  runtime = clock() - runtime;
  printf("For %u x %u Matrix\n",n,n);
  cout << "Runtime for LU Decomposition is: " << (runtime)/float(CLOCKS_PER_SEC) << endl;
  
  double** A = new double* [n];
  double** u = new double* [n];
  double** l = new double* [n];
 
  
  initialize_matrices(A,u,l,n);
  for(int i = 0 ;i < n ; ++i){
    for(int j= 0; j < n; ++j){
      A[i][j]=ret[i*n+j];
    }
  }
  
  for(int i=0; i<n; i++){
    for(int j=0; j<n; j++){
        
        for(int k=0; k<n; k++){
            
            if(i>=k)
              l[i][k] = A[i][k];
            
            else l[i][k] = 0;
            
            if(k==j)u[k][j] = 1;
            else if(k<j)u[k][j] = A[k][j];
            else u[k][j] = 0.0;
        }
    }
  }
  
 if(atoi(argv[2]) == 1){
  printf("Matrix 'A' is:\n");
  printMatrix(a,n);  
  printf("Matrix 'L' is:\n");
  print2D(l,n);
  printf("Matrix 'U' is: \n");
  print2D(u,n);
 }


  hipFree(da);
  hipFree(ret);
  delete[] a;
  delete[] ret; 

  return 0;
}